#include "hip/hip_runtime.h"
#include "cuda_globals.h"

constexpr static double PI = 3.141592653589793;
static hipEvent_t custart, cuend;
// ---

template<typename T>
__device__
unsigned unmap( T v, const T v0, const T v1, const T len)
{
  return static_cast<unsigned>( (v-v0)/(v1-v0)*len );
}

template<typename T>
__device__
T map( unsigned v, const T v0, const T v1, const T len)
{
  return static_cast<T>(v)/len*(v1-v0)+v0;
}


__device__ inline
unsigned char toColor(float v) {
  return static_cast<unsigned char>(255.0f*saturate(v));
}


/// HSL [0:1] to RGB {0..255}, from http://stackoverflow.com/questions/4728581/hsl-image-adjustements-on-gpu
__device__
void hsl2rgb( float hue, float sat, float lum, uchar4& color )
{
  const float onethird = 1.0 / 3.0;
  const float twothird = 2.0 / 3.0;
  const float rcpsixth = 6.0;

  float xtr = rcpsixth * (hue - twothird);
  float xtg = 0.0;
  float xtb = rcpsixth * (1.0 - hue);

  if (hue < twothird) {
    xtr = 0.0;
    xtg = rcpsixth * (twothird - hue);
    xtb = rcpsixth * (hue      - onethird);
  }

  if (hue < onethird) {
    xtr = rcpsixth * (onethird - hue);
    xtg = rcpsixth * hue;
    xtb = 0.0;
  }

  xtr = __saturatef(xtr);
  xtg = __saturatef(xtg);
  xtb = __saturatef(xtb);

  float sat2   =  2.0 * sat;
  float satinv =  1.0 - sat;
  float luminv =  1.0 - lum;
  float lum2m1 = (2.0 * lum) - 1.0;
  float ctr    = (sat2 * xtr) + satinv;
  float ctg    = (sat2 * xtg) + satinv;
  float ctb    = (sat2 * xtb) + satinv;

  if (lum >= 0.5) {
    color.x = toColor((luminv * ctr) + lum2m1);
    color.y = toColor((luminv * ctg) + lum2m1);
    color.z = toColor((luminv * ctb) + lum2m1);
  }else {
    color.x = toColor(lum * ctr);
    color.y = toColor(lum * ctg);
    color.z = toColor(lum * ctb);
  }
}


template<typename T>
__global__
void d_init_buffer(
  Data<T> _data,
  const Parameters<T> _params)
{
  unsigned i,j,offset_ij;
  T width = _params.width;
  T height = _params.height;
  for (i = blockIdx.y * blockDim.y + threadIdx.y;
       i < _params.height;
       i += blockDim.y * gridDim.y)
  {
    for (j = blockIdx.x * blockDim.x + threadIdx.x;
         j < _params.width;
         j += blockDim.x * gridDim.x)
    {
      offset_ij = j+i*_params.width;
      _data.buffer[offset_ij]             = 0.0f;
      _data.buffer[offset_ij+1*_params.n] = 0.0f;
      _data.buffer[offset_ij+2*_params.n] = 0.0f;
      _data.buffer[offset_ij+3*_params.n] = map(j, _params.x0, _params.x1, width);
      _data.buffer[offset_ij+4*_params.n] = map(i, _params.y0, _params.y1, height);
    }
  }
}

/**
 *
 */
__global__ void d_clear_color(uchar4 *ptr, unsigned n)
{
  unsigned i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n)
    return;
  ptr[i].x = 0;
  ptr[i].y = 0;
  ptr[i].z = 0;
}

template<unsigned TFuncId, typename T>
inline __device__
T funcX(T t, const T time, T xk, T yk, const Parameters<T>& params) {
  switch(TFuncId) {
  case 0: return xk+params.talpha*cos(params.t0+time+yk+cos(params.t1+time+PI*xk));
  case 1: return t*yk+0.95f*xk-params.talpha*sin(0.7f*yk);//+sin(3.0f*0.7f*yk));
    // xk-hf(y+hf(x)), f(x)=sin(x+sin(3x))
  case 2: return xk-params.talpha*sin( yk+params.talpha*sin(params.t0+time+xk+sin(params.t1+time+3.0*xk)) + sin(3.0*(yk+params.talpha*sin(params.t0+time+xk+sin(params.t1+time+3.0*xk)))) );
  case 3: return xk-params.talpha*sin( params.t0+yk+time+sin(3*yk+params.t1+time+sin(2*yk+time)) );
  }
  return xk;
}

template<unsigned TFuncId, typename T>
inline __device__
T funcY(T t, const T time, T xk, T yk, const Parameters<T>& params) {
  switch(TFuncId) {
  case 0: return yk+params.talpha*cos( params.t2+time+xk+cos(params.t3+time+PI*yk));
  case 1: return t*xk+0.95f*yk+params.talpha*sin(0.6f*xk);//+sin(3.0f*0.6f*xk));
  case 2: return yk+params.talpha*sin(params.t2+time+xk+sin(params.t3+time+3.0*xk));
  case 3: return yk+params.talpha*sin( xk+params.t2+time+sin(3*xk+params.t3+time+sin(2*xk+time)) );
  }
  return yk;
}

template<unsigned TFuncId, bool TColoring, typename T>
__global__
void d_generate_pattern(
  Data<T> _data,
  const Parameters<T> _params,
  const T _iteration_start,
  const T _iteration_end,
  const T _iteration_step_size
  )
{
  unsigned i,j;
  unsigned px, py;
  unsigned offset_ij;
  T xk,yk;
  T width = _params.width;
  T height = _params.height;
  T t;
  for (i = blockIdx.y * blockDim.y + threadIdx.y;
       i < _params.height;
       i += blockDim.y * gridDim.y)
  {
    for (j = blockIdx.x * blockDim.x + threadIdx.x;
         j < _params.width;
         j += blockDim.x * gridDim.x)
    {
      offset_ij = j+i*_params.width;
      xk = _data.buffer[offset_ij + 3*_params.n];
      yk = _data.buffer[offset_ij + 4*_params.n];

      for(t=_iteration_start; t<_iteration_end; t+=_iteration_step_size) {
        xk = funcX<TFuncId>(t, _params.time, xk, yk, _params);
        yk = funcY<TFuncId>(t, _params.time, xk, yk, _params);
        px = unmap(xk, _params.x0, _params.x1, width);
        py = unmap(yk, _params.y0, _params.y1, height);
        if (px<_params.width && py<_params.height) {
          unsigned offset = px+py*_params.width;
          T v = _params.addValue*powf(1.0f-t, _params.density_slope);
          if(_params.use_atomics)
            atomicAdd(_data.buffer+offset, v); // just density
          else
            _data.buffer[offset] += v;
        }
      } // for
      _data.buffer[offset_ij + 3*_params.n] = xk;
      _data.buffer[offset_ij + 4*_params.n] = yk;
    }
  }
}

template<bool TColoring, typename T>
__global__
void d_render_to_image(
  uchar4 *_ptr,
  Data<T> _data,
  const Parameters<T> _params
  )
{
  unsigned j;
  for (j = blockIdx.x * blockDim.x + threadIdx.x;
       j < _params.n;
       j += blockDim.x * gridDim.x)
  {
    if(TColoring)
    {
      T v = _data.buffer[j];
      float h;
      float s;
      float l;
      if(_params.hue_end>=_params.hue_start)
        h = powf((_params.hue_end-_params.hue_start)*__saturatef(v), _params.hue_slope) + _params.hue_start;
      else
        h = powf((_params.hue_start-_params.hue_end)*(__saturatef(v)), _params.hue_slope) + _params.hue_end;
      s = __saturatef(powf(v, _params.saturation_slope));
      l = __saturatef(powf(v, _params.brightness_slope));
      if(_params.invert)
        l = 1.0f-l;
      hsl2rgb(h, s, l, _ptr[j]);
    }else{
      T density = sqrt(_data.buffer[j]);//exp(-data.buffer[j])
      _ptr[j].x = toColor( 1.0-0.3*powf(density,0.4) );
      _ptr[j].y = toColor( 1.0-0.5*powf(density,1.0) );
      _ptr[j].z = toColor( 1.0-0.8*powf(density,1.4) );
      /*unsigned char d = 255*data.buffer[j];//exp(-data.buffer[j])
        ptr[j].x = d;
        ptr[j].y = d;
        ptr[j].z = d;*/
    }
  }
}
/**
 *
 */
template<unsigned TFuncId, bool TColoring, typename T>
float launch_kernel(
  hipGraphicsResource* _dst,
  Data<T>& _ddata,
  const Parameters<T>& _params,
  unsigned _iteration_offset)
{
  int numSMs;
  int devId = 0;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);

  dim3 threads( 16, 16 );
  dim3 threads1d( 128 );
  dim3 blocks( 32*numSMs );
  size_t num_bytes;
  hipError_t err;
  float ms = 0.0f;

  err=hipGraphicsMapResources(1, &_dst, 0);
  if (err == hipSuccess)  {
    uchar4* pos;
    const T it_start = _iteration_offset/T(_params.max_iterations);
    const T it_end = it_start + _params.iterations_per_run/T(_params.max_iterations);
    const T it_step_size = 1.0/T(_params.max_iterations);

    CHECK_CUDA(hipGraphicsResourceGetMappedPointer(
                 (void**)&pos, &num_bytes, _dst));

    CHECK_CUDA(hipEventRecord(custart));
    if(_iteration_offset==0) {
      d_generate_pattern<TFuncId, TColoring>
        <<<blocks, threads>>>(_ddata, _params, it_start, it_end, it_step_size);
    }else{
      d_generate_pattern<TFuncId, TColoring>
        <<<blocks, threads>>>(_ddata, _params, it_start, it_end, it_step_size);
    }

    CHECK_CUDA(hipEventRecord(cuend));

    d_render_to_image<TColoring><<<blocks, threads1d>>>(pos, _ddata, _params);

    CHECK_CUDA( hipEventSynchronize(cuend) );
    CHECK_CUDA( hipEventElapsedTime(&ms, custart, cuend) );
  }
  CHECK_CUDA( hipGraphicsUnmapResources(1, &_dst, 0));
  return ms;
}

/**
 *
 */
template<typename T>
void alloc_buffer(
  Data<T>& ddata,
  const Parameters<T>& params)
{
  if(ddata.buffer) {
    CHECK_CUDA( hipFree(ddata.buffer) );
    CHECK_CUDA( hipEventDestroy(custart) );
    CHECK_CUDA( hipEventDestroy(cuend) );
  }
  unsigned n = 5 * params.n;
  CHECK_CUDA( hipMalloc(&ddata.buffer, n*sizeof(T)) );
  CHECK_CUDA( hipEventCreate(&custart) );
  CHECK_CUDA( hipEventCreate(&cuend) );
}

/**
 *
 */
template<typename T>
void init_buffer(
  Data<T>& ddata,
  const Parameters<T>& params)
{
  //unsigned n = 5 * params.n;
  //  CHECK_CUDA( hipMemset(ddata.buffer, 0.0, n*sizeof(T)));
  int numSMs;
  int devId = 0;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);

  dim3 threads( 16, 16 );
  dim3 blocks( 32*numSMs );
  d_init_buffer<<<blocks, threads>>>(ddata, params);
  CHECK_CUDA( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );
}

/**
 *
 */
template<typename T>
void cleanup_cuda(Data<T>& ddata)
{
  if(ddata.buffer) {
    CHECK_CUDA( hipFree(ddata.buffer) );
    ddata.buffer = 0;
  }
}


template
void alloc_buffer<float>(Data<float>&, const Parameters<float>&);
template
void init_buffer<float>(Data<float>&, const Parameters<float>&);
template float launch_kernel<0, false, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<1, false, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<2, false, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<3, false, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<0, true, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<1, true, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<2, true, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template float launch_kernel<3, true, float>(hipGraphicsResource*, Data<float>&, const Parameters<float>&, unsigned);
template
void cleanup_cuda<float>(Data<float>&);

/*
template
void alloc_buffer<double>(Data<double>&, const Parameters<double>&);
template
void init_buffer<double>(Data<double>&, const Parameters<double>&);
template float launch_kernel<0, false, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<1, false, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<2, false, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<3, false, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<0, true, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<1, true, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<2, true, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template float launch_kernel<3, true, double>(hipGraphicsResource*, Data<double>&, const Parameters<double>&);
template
void cleanup_cuda<double>(Data<double>&);
*/
