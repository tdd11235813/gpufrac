#include "hip/hip_runtime.h"
#include "cuda_globals.hpp"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <hiprand/hiprand_kernel.h>
/// @todo : variable seed, const restrict, grid-strides
// ---
#include <cudpp.h>


template<typename T>
void cinit(DataMc<T>&, uint width, uint height);
template<typename T>
void cfin(DataMc<T>&);
template<typename T>
void blur_sat(DataMc<T>& _data,
              T* target,
              T* backBuffer,
              T* source,
              const Parameters<T>& );

hipEvent_t cstart, cend;
size_t d_satPitch = 0;
size_t d_satPitch_T = 0;
size_t d_satPitchInElements = 0;
size_t d_satPitchInElements_T = 0;
CUDPPHandle theCudpp;
CUDPPHandle scanPlan;
CUDPPConfiguration config = { CUDPP_SCAN,
                              CUDPP_ADD,
                              CUDPP_FLOAT, // @todo
                              CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE };
hiprandStatePhilox4_32_10_t *devStates;

// ---
// @todo T
static constexpr float dsinus[]   = {0,0.0, 0.0,0.866025,1,0.951057,0.866025};
static constexpr float dcosinus[] = {0,1.0,-1.0,-0.5,0,0.309017,0.5};


__device__ inline
unsigned char toColor(float v) {
  return static_cast<unsigned char>(255.0f*__saturatef(v));
}

/// HSL [0:1] to RGB {0..255}, from http://stackoverflow.com/questions/4728581/hsl-image-adjustements-on-gpu
__device__
void hsl2rgb_mccabe( float hue, float sat, float lum, uchar4& color )
{
  const float onethird = 1.0 / 3.0;
  const float twothird = 2.0 / 3.0;
  const float rcpsixth = 6.0;

  float xtr = rcpsixth * (hue - twothird);
  float xtg = 0.0;
  float xtb = rcpsixth * (1.0 - hue);

  if (hue < twothird) {
    xtr = 0.0;
    xtg = rcpsixth * (twothird - hue);
    xtb = rcpsixth * (hue      - onethird);
  }

  if (hue < onethird) {
    xtr = rcpsixth * (onethird - hue);
    xtg = rcpsixth * hue;
    xtb = 0.0;
  }

  xtr = __saturatef(xtr);
  xtg = __saturatef(xtg);
  xtb = __saturatef(xtb);

  float sat2   =  2.0 * sat;
  float satinv =  1.0 - sat;
  float luminv =  1.0 - lum;
  float lum2m1 = (2.0 * lum) - 1.0;
  float ctr    = (sat2 * xtr) + satinv;
  float ctg    = (sat2 * xtg) + satinv;
  float ctb    = (sat2 * xtb) + satinv;

  if (lum >= 0.5) {
    color.x = toColor((luminv * ctr) + lum2m1);
    color.y = toColor((luminv * ctg) + lum2m1);
    color.z = toColor((luminv * ctb) + lum2m1);
  }else {
    color.x = toColor(lum * ctr);
    color.y = toColor(lum * ctg);
    color.z = toColor(lum * ctb);
  }
}


template<typename T>
inline void find_min_max(T* begin, T* end, T *min, T *max){
    thrust::pair< thrust::device_ptr<T>, thrust::device_ptr<T> > tuple;
    tuple =
        thrust::minmax_element(
          thrust::device_ptr<T>(begin),
          thrust::device_ptr<T>(end)
        );

    *min = tuple.first[0];
    *max = tuple.second[0];
}


template<typename TRandState>
__global__ void d_setup_kernel(TRandState *state, uint n, int seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id>=n)
      return;
    hiprand_init(seed, id, 0, state+id);
}

/**
 * @todo to 4dim vec
 */
template<typename T, typename TRandState>
__global__ void d_initialize4( DataMc<T> _data,
                               const Parameters<T> _params,
                               TRandState *state,
                               uint n)
{
  using TVec = typename std::conditional<std::is_same<T,float>::value,float4,double4>::type;

  uint i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n)
    return;
  uint iy = i / _params.height;
  TRandState localState = state[i];
  TVec* pgrid4 = reinterpret_cast<TVec*>(_data.grid);
  TVec val4 = hiprand_uniform4(&localState);
  T gradient = static_cast<T>(iy)/_params.height;
  val4.x = (2.0*val4.x-1.0)*gradient;
  val4.y = (2.0*val4.y-1.0)*gradient;
  val4.z = (2.0*val4.z-1.0)*gradient;
  val4.w = (2.0*val4.w-1.0)*gradient;
  pgrid4[i] = val4;
  state[i] = localState;
}
/**
 *
 */
/*template<typename T, int TINVERT_MODE>
__global__ void d_reset_pattern( DataMc<T> data, hiprandState *state, int radius )
{
  int ix = (threadIdx.x + blockIdx.x * blockDim.x);
  int iy = (threadIdx.y + blockIdx.y * blockDim.y);
  int i = ix + iy * blockDim.x * gridDim.x;
//  uint i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= _params.n)
    return;
  if (ix<20 || iy<20 || ix+20>_params.width || iy+20>_params.height)
    return;
  hiprandState localState = state[i];
  T rnd = (T)hiprand_uniform(&localState);
  T gm = TINVERT_MODE==1?-1.:1.;
  if(rnd<T(0.0007)){
    for(int x=-radius; x<=radius; ++x)
      for(int y=-radius; y<=radius; ++y)
      {
        if(sqrtf(x*x+y*y)<float(radius))
          data.grid[ix+x + (iy+y)*_params.width] = gm;
      }
  }
  //state[i] = localState;
  }*/
/**
 *
 */
 /*template<typename T>
__global__ void d_clear_color(uchar4 *ptr, const Parameters<T> _params)
{
  uint i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= _params.n)
    return;
  ptr[i].x = 0;
  ptr[i].y = 0;
  ptr[i].z = 0;
}*/
/**
 *
 */
template<unsigned TSymmetry, typename T>
__device__ int getSymmetry(int i, int ix, int iy, const Parameters<T>& _params)
{
  if(TSymmetry == 2)
    return _params.n - 1 - i;
  else {
    float dx = ix - int(_params.width>>1);
    float dy = iy - int(_params.height>>1);
    int x2 = int(_params.width>>1)  + (int)(dx * dcosinus[TSymmetry] + dy * dsinus[TSymmetry]);
    int y2 = int(_params.height>>1) + (int)(dx * -dsinus[TSymmetry] + dy * dcosinus[TSymmetry]);
    int j = x2 + y2 * int(_params.width);

    return j<0 ? j+_params.n : j>=_params.n ? j-_params.n : j;
  }
}
/**
 *
 */
template<unsigned TSymmetry, typename T>
__global__ void d_symmetry(DataMc<T> _data, const Parameters<T> _params)
{
  uint ix = (threadIdx.x + blockIdx.x * blockDim.x);
  uint iy = (threadIdx.y + blockIdx.y * blockDim.y);
  uint i = ix + iy * _params.width;
  if (ix >= _params.width || iy >= _params.height)
    return;
  uint index = getSymmetry<TSymmetry>(i, ix, iy, _params);
  _data.grid[i] = _data.grid[i] * .94 + _data.backBuffer[index] * .06;
}
/**
 *
 */
template<typename T>
__global__ void d_blur_step2(T* target, T* backBuffer, T* sat,
                             size_t pitch, T* source, const Parameters<T> _params)
{
  uint ix = (threadIdx.x + blockIdx.x * blockDim.x);
  uint iy = (threadIdx.y + blockIdx.y * blockDim.y);
  uint i = ix + iy * _params.width;
  if (ix>=_params.width || iy>=_params.height)
    return;
  if(ix==0)
    target[i] = backBuffer[iy*_params.width] + source[iy*_params.width];
  else if(iy==0)
    target[i] = backBuffer[ix] + source[ix];
  else
  {
    target[i] = backBuffer[ix]
      + backBuffer[iy*_params.width]
      - backBuffer[0]
      + sat[ix + pitch*iy]
      -sat[(iy-1)*pitch]
      -sat[ix-1];
  }
}
/**
 *
 */
template<typename T>
__global__ void d_collect(T* to, T* buffer, uint radius, const Parameters<T> _params)
{
  uint ix = (threadIdx.x + blockIdx.x * blockDim.x);
  uint iy = (threadIdx.y + blockIdx.y * blockDim.y);
  uint i = ix + iy * _params.width;
//  uint i = ix + iy * _params.width;
  if (ix>=_params.width || iy>=_params.height)
    return;
  int minx = ix>radius ? ix-radius : 0;
  int maxx = min(ix + radius, _params.width - 1);
  int miny = iy>radius ? iy-radius : 0;
  int maxy = min(iy + radius, _params.height - 1);
  T area = 1.0/static_cast<T>((maxx - minx) * (maxy - miny));
  to[i] = ( buffer[maxy * _params.width + maxx]
            - buffer[maxy * _params.width + minx]
            - buffer[miny * _params.width + maxx]
            + buffer[miny * _params.width + minx]) * area;
}

#define M_GETBEST_J(elem, kk) if (TIsLevelZero || vabs.elem<var.elem) { \
        var4[j].elem = vabs.elem;\
        lvl4[j].elem = level;\
        dir4[j].elem = src.elem>tgt.elem;\
      }
/**
 *
 */
template<typename T, bool TIsLevelZero>
__global__ void d_getBest(DataMc<T> _data, T* target, T* source, int level, unsigned n)
{
  unsigned i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < n;
       i += blockDim.x * gridDim.x)
  {
    T variation = fabs(source[i]-target[i]);
    if (TIsLevelZero || variation < _data.bestVariation[i]) {
      _data.bestVariation[i] = variation;
      _data.bestLevel[i] = level;
      _data.direction[i] = source[i] > target[i];
    }
  }
}

/**
 * @tparam TDir 0==default, 1==neg.dir, 2==pos.dir
 */
template<typename T, int TDir>
__global__ void d_advance(DataMc<T> _data, const Parameters<T> _params)
{
  int i;
  T delta = 100.0*_params.time_delta;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _params.n;
       i += blockDim.x * gridDim.x)
  {
      T curStep = delta*_data.stepSizes[_data.bestLevel[i]];
      if (TDir==1 || (!_data.direction[i] && TDir!=2) )
      {
        curStep = -curStep;
      }
      _data.grid[i] += curStep;
      _data.colorgrid[i] += curStep * _data.colorShift[_data.bestLevel[i]];
  }
}

template<typename T>
__global__ void d_dumpToImage(
      uchar4 *ptr,
      T* buffer,
      const Parameters<T> _params)
{
  uint i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= _params.n)
    return;
  ptr[i].x = 255*buffer[i];
}

template<bool TInvert, typename T>
__global__ void d_renderPattern(
      uchar4 *ptr,
      const Parameters<T> _params,
      T* grid,
      T* colorgrid,
      T gridmin,
      T gridrange,
      T colormin,
      T colorrange
    )
{
  uint i=threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= _params.n)
    return;
  float v,h,s,l;

  grid[i] = ((grid[i]-gridmin)/gridrange)-1.0;
  colorgrid[i] = ((colorgrid[i]-colormin)/colorrange)-1.0;

  if(TInvert)
  {
    v = 0.5f - 0.5f*grid[i];
    h = 0.5f - 0.5f*colorgrid[i];
  }else{
    v = 0.5f*grid[i]+0.5f;
    h = 0.5f*colorgrid[i]+0.5f;
  }
//(dmap(colorgrid[i], gmin, gmax, static_cast<float>(_params.hue_start), static_cast<float>(_params.hue_end)));
//  float v = dmap(grid[i], gmin, gmax, 0.0f, 1.0f);

//  int hue_offset = 255.0f * (_params.hue_start < 0.0 ? _params.hue_start+1.0 : _params.hue_start);

  v *= _params.density_slope;
  if(_params.hue_end>=_params.hue_start)
    h = powf((_params.hue_end-_params.hue_start)*__saturatef(h), _params.hue_slope) + _params.hue_start;
  else
    h = powf((_params.hue_start-_params.hue_end)*(__saturatef(h)), _params.hue_slope) + _params.hue_end;
  if(h<0.0f)
    h += 1.0f;
  else if(h>1.0f)
    h -= 1.0f;

  s = __saturatef(powf(v, _params.saturation_slope));
  l = __saturatef(powf(v, _params.brightness_slope));
  hsl2rgb_mccabe(h,s,l,ptr[i]);
}

/**
 *
 */
template<typename T>
float launch_kernel(hipGraphicsResource* dst,
                    DataMc<T>& _data,
                    const Parameters<T>& _params,
                    bool advance,
                    int direction_mode)
{
  uchar4* pos;
  size_t num_bytes;
  uint radius;
  hipError_t err;
  float ms = 0.0f;
  int numSMs;

  T* backbuffer = _data.backBuffer;
  T* grid = _data.grid;
  T* blurbuffer = _data.blurBuffer;
  T* diffusion_right = _data.diffusionRight;
  T* diffusion_left  = _data.diffusionLeft;
  T* colorgrid  = _data.colorgrid;
  T* source = grid;
  T* target = diffusion_right;

  T gridmin, gridmax, gridrange;
  T colormin, colormax, colorrange;

  err=hipGraphicsMapResources(1, &dst, 0);
  if (err == hipSuccess)
  {
    CHECK_CUDA(hipGraphicsResourceGetMappedPointer(
        (void**)&pos, &num_bytes, dst));

    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    dim3 threads_1(128);
    dim3 blocks_1( (_params.n-1)/threads_1.x+1 );
    dim3 threads_2(32, 4);
    dim3 blocks_2((_params.width-1)/threads_2.x+1, (_params.height-1)/threads_2.y+1);
    dim3 blocks_1_sm( 32*numSMs );

    //d_clear_color<<<blocks_1,threads_1>>>(pos);

    /*
    int kradius;
    static long long unsigned ff = 0;
    ff += 1;
    if((ff&15)==15)
    {
      kradius = ff/250;
      if(kradius>5)
        kradius = 1;
      else
        kradius = 5-kradius;
      if(ff>7*250)
        ff = 0;
      if(M_CONTAINS_FLAG_TRUE(mode,M_INVERT_MODE))
        d_reset_pattern<T,1><<<blocks_2, threads_2>>>(*_data, devStates, kradius );
      else
        d_reset_pattern<T,0><<<blocks_2, threads_2>>>(*_data, devStates, kradius );
    }*/

    CHECK_CUDA(hipEventRecord(cstart));

    if(advance) {
      if(_data.symmetry>0)
      {
        CHECK_CUDA(hipMemcpy(backbuffer, grid, _params.n*sizeof(T), hipMemcpyDeviceToDevice));
        switch(_data.symmetry) {
        case 1: d_symmetry<2><<<blocks_2, threads_2>>>(_data, _params); break;
        case 2: d_symmetry<3><<<blocks_2, threads_2>>>(_data, _params); break;
        case 3: d_symmetry<4><<<blocks_2, threads_2>>>(_data, _params); break;
        case 4: d_symmetry<5><<<blocks_2, threads_2>>>(_data, _params); break;
        }
      }

      for (int level = 0; level < _data.levels; level++)
      {
        radius = _data.radii_host[level];
        CHECK_CUDA(hipMemcpy(backbuffer, blurbuffer, _params.n*sizeof(T), hipMemcpyDeviceToDevice));
        if(level<=_data.blurlevels){
          blur_sat(_data, blurbuffer, backbuffer, source, _params);
        }

        d_collect<T><<<blocks_2,threads_2>>>(target, blurbuffer, radius, _params);
        if(level==0)
          d_getBest<T,true><<<blocks_1_sm, threads_1>>>(_data, target, source, level, _params.n);
        else
          d_getBest<T,false><<<blocks_1_sm, threads_1>>>(_data, target, source, level, _params.n);
        //        d_dumpToImage<<<blocks_1, threads_1>>>(pos, blurbuffer, _params);
        if((level&1)==0)
        {
          source = target;
          target = diffusion_left;
        }else{
          source = target;
          target = diffusion_right;
        }
      } // level


      if(direction_mode==0)
        d_advance<T,0><<<blocks_1_sm, threads_1>>>(_data, _params);
      else if(direction_mode==1)
        d_advance<T,1><<<blocks_1_sm, threads_1>>>(_data, _params);
      else
        d_advance<T,2><<<blocks_1_sm, threads_1>>>(_data, _params);

      find_min_max(grid, grid+_params.n, &gridmin, &gridmax);
      gridrange = 0.5*(gridmax - gridmin);
      find_min_max(colorgrid, colorgrid+_params.n, &colormin, &colormax);
      colorrange = 0.5*(colormax - colormin);
    }else{
      gridmin = -1.f;
      gridrange = 1.f;
      colormin = -1.f;
      colorrange = 1.f;
    }

    if(_params.invert)
      d_renderPattern<true>
        <<<blocks_1, threads_1>>>(pos, _params, grid, colorgrid, gridmin, gridrange, colormin, colorrange);
    else
      d_renderPattern<false>
        <<<blocks_1, threads_1>>>(pos, _params, grid, colorgrid, gridmin, gridrange, colormin, colorrange);

    CHECK_CUDA( hipEventRecord(cend) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipEventElapsedTime(&ms, cstart, cend) );
    CHECK_CUDA( hipGraphicsUnmapResources(1, &dst, 0));
  }

  return ms;
}

/**
 *
 */
template<typename T>
void init_buffer(DataMc<T>& _data,
                 const Parameters<T>& _params,
                 bool alloc,
                 int seed)
{
  if(_data.base<=1.0)
    throw std::runtime_error("McCabe: invalid base value (must be > 1.0)");

  if(alloc)
  {
    if(_data.backBuffer)
    {
      CHECK_CUDA( hipFree(_data.backBuffer) );
      CHECK_CUDA( hipFree(_data.grid) );
      CHECK_CUDA( hipFree(_data.diffusionLeft) );
      CHECK_CUDA( hipFree(_data.diffusionRight) );
      CHECK_CUDA( hipFree(_data.blurBuffer) );
      CHECK_CUDA( hipFree(_data.bestVariation) );
      CHECK_CUDA( hipFree(_data.colorgrid) );
      CHECK_CUDA( hipFree(_data.bestLevel) );
      CHECK_CUDA( hipFree(_data.direction) );
      CHECK_CUDA(hipFree(devStates));
      CHECK_CUDA(hipEventDestroy(cstart));
      CHECK_CUDA(hipEventDestroy(cend));
      cfin(_data);
    }

    CHECK_CUDA(hipEventCreate(&cstart));
    CHECK_CUDA(hipEventCreate(&cend));

    CHECK_CUDA( hipMalloc(&_data.backBuffer, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.grid, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.diffusionLeft, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.diffusionRight, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.blurBuffer, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.bestVariation, _params.n*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.colorgrid, _params.n*sizeof(T)) );

    CHECK_CUDA( hipMalloc(&_data.bestLevel, _params.n*sizeof(int)) );
    CHECK_CUDA( hipMemset(_data.bestLevel, 0, _params.n*sizeof(int)));

    CHECK_CUDA( hipMalloc(&_data.direction, _params.n*sizeof(bool)) );
    CHECK_CUDA( hipMemset(_data.direction, 0, _params.n*sizeof(bool)));

    CHECK_CUDA( hipMemset(_data.blurBuffer, 0.0, _params.n*sizeof(T)));
    CHECK_CUDA( hipMemset(_data.diffusionLeft, 0.0, _params.n*sizeof(T)));
    CHECK_CUDA( hipMemset(_data.diffusionRight, 0.0, _params.n*sizeof(T)));
    CHECK_CUDA( hipMemset(_data.bestVariation, 0.0, _params.n*sizeof(T)));
    CHECK_CUDA( hipMemset(_data.colorgrid, 0.0, _params.n*sizeof(T)));

    CHECK_CUDA(hipMalloc(&devStates, _params.n * sizeof(hiprandStatePhilox4_32_10_t)));

  }

  int radius;
  // Pos Most Sig Bit - 1
  int new_levels = (int) (logf(max(_params.width,_params.height)) / logf(_data.base)) - 1;
  int new_blurlevels = (int) ((_data.levels+1.0f) * _data.blurFactor - 0.5f);
  if(new_blurlevels<0)
    new_blurlevels=0;

  if(_data.levels != new_levels)
  {
    _data.levels     = new_levels;
    _data.blurlevels = new_blurlevels;

    delete[] _data.radii_host;
    CHECK_CUDA( hipFree(_data.radii) );
    CHECK_CUDA( hipFree(_data.stepSizes) );
    CHECK_CUDA( hipFree(_data.colorShift) );

    _data.radii_host = new unsigned[_data.levels];
    CHECK_CUDA( hipMalloc(&_data.radii, _data.levels*sizeof(unsigned)) );
    CHECK_CUDA( hipMalloc(&_data.stepSizes, _data.levels*sizeof(T)) );
    CHECK_CUDA( hipMalloc(&_data.colorShift, _data.levels*sizeof(T)) );

  }

  auto* stepSizes   = new T[_data.levels];
  auto* colorShift  = new T[_data.levels];

  for(int i=0; i<_data.levels; ++i)
  {
    radius = (uint) pow(_data.base, i);
    _data.radii_host[i] = radius;
    stepSizes[i] = log(radius) * _data.stepScale + _data.stepOffset;
    colorShift[i] = ((i & 0x01) == 0 ? -1.0 : 1.0) * (_data.levels-i);
    //printf("i %d: r=%.3f, s=%.3f, c=%.3f\n",i, temps.radii[i], temps.stepSizes[i], temps.colorShift[i]);
    CHECK_CUDA(hipMemcpy(_data.radii, _data.radii_host, _data.levels*sizeof(uint), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(_data.stepSizes, stepSizes, _data.levels*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(_data.colorShift, colorShift, _data.levels*sizeof(T), hipMemcpyHostToDevice));
  }
  delete[] stepSizes;
  delete[] colorShift;

  if(alloc) {
    uint n = (_params.n+3)/4;
    uint threads_1 = 128;
    uint blocks_1 = (n-1)/threads_1+1;
    d_setup_kernel<<<blocks_1, threads_1>>>(devStates, n, seed);
    d_initialize4<T><<<blocks_1,threads_1>>>(_data, _params, devStates, n);
    CHECK_LAST("Initialization failed.");

    cinit(_data, _params.width, _params.height);
  }
}
/**
 *
 */
template<typename T>
void cleanup_cuda(DataMc<T>& _data)
{
  if(_data.backBuffer)
  {
    CHECK_CUDA( hipFree(_data.backBuffer) );
    CHECK_CUDA( hipFree(_data.grid) );
    CHECK_CUDA( hipFree(_data.diffusionLeft) );
    CHECK_CUDA( hipFree(_data.diffusionRight) );
    CHECK_CUDA( hipFree(_data.blurBuffer) );
    CHECK_CUDA( hipFree(_data.bestVariation) );
    CHECK_CUDA( hipFree(_data.colorgrid) );
    CHECK_CUDA( hipFree(_data.bestLevel) );
    CHECK_CUDA( hipFree(_data.direction) );
    CHECK_CUDA(hipEventDestroy(cstart));
    CHECK_CUDA(hipEventDestroy(cend));

    CHECK_CUDA(hipFree(devStates));
    _data.backBuffer = nullptr;
    cfin(_data);
  }
  if(_data.radii)
  {
    delete[] _data.radii_host;
    CHECK_CUDA( hipFree(_data.radii) );
    CHECK_CUDA( hipFree(_data.stepSizes) );
    CHECK_CUDA( hipFree(_data.colorShift) );
    _data.radii = nullptr;
    _data.stepSizes = nullptr;
    _data.colorShift = nullptr;
    _data.radii_host = nullptr;
  }
  _data.levels = -1;
  _data.symmetry = 0;
}

// ---
// http://stackoverflow.com/questions/14174876/cuda-in-place-transpose-error
template<typename T, int TBlockSize>
__global__ void d_transpose(T* dst, T* src, int dstPitch, int srcPitch, int width, int height)
{
  __shared__ T block[TBlockSize][TBlockSize];

  int col = blockIdx.x * TBlockSize + threadIdx.x;
  int row = blockIdx.y * TBlockSize + threadIdx.y;

  if((col < width) && (row < height))
  {
    int tid_in = row * srcPitch + col;
    block[threadIdx.y][threadIdx.x] = src[tid_in];
  }

  __syncthreads();

  col = blockIdx.y * TBlockSize + threadIdx.x;
  row = blockIdx.x * TBlockSize + threadIdx.y;

  if((col < height) && (row < width))
  {
    int tid_out = row * dstPitch + col;
    dst[tid_out] = block[threadIdx.x][threadIdx.y];
  }
}

template<typename T>
void cinit(DataMc<T>& _data, uint width, uint height)
{
    size_t dpitch   = width  * sizeof(T);
    size_t dpitch_T = height * sizeof(T);

    CHECK_LAST("Before CUDA initialization");

    CHECK_CUDA( hipMallocPitch( _data.SATs, &d_satPitch, dpitch, height));
    CHECK_CUDA( hipMallocPitch( _data.SATs+1, &d_satPitch_T, dpitch_T, width));

    d_satPitchInElements   = d_satPitch   / sizeof(T);
    d_satPitchInElements_T = d_satPitch_T / sizeof(T);
    // Initialize CUDPP
    cudppCreate(&theCudpp);
}

template<typename T>
void cfin(DataMc<T>& _data)
{
  CHECK_CUDA(hipFree(_data.SATs[0]));
  CHECK_CUDA(hipFree(_data.SATs[1]));
  _data.SATs[0] = nullptr;
  _data.SATs[1] = nullptr;
  // shut down CUDPP
  if (CUDPP_SUCCESS != cudppDestroy(theCudpp))
  {
      printf("Error destroying CUDPP.\n");
  }
}

template<typename T>
void blur_sat(DataMc<T>& _data,
              T* _target,
              T* _backBuffer,
              T* _source,
              const Parameters<T>& _params)
{
  dim3 threads_2(16, 16);
  dim3 blocks_2  ((_params.width-1) / threads_2.x+1,
                  (_params.height-1) / threads_2.y+1);
  dim3 blocks_2_T((_params.height-1) / threads_2.x+1,
                  (_params.width-1) / threads_2.y+1);

  CHECK_CUDA(
      hipMemcpy2D(_data.SATs[0], d_satPitch, _source, _params.width * sizeof(T),
                   _params.width * sizeof(T), _params.height,
                   hipMemcpyDeviceToDevice));

  if (CUDPP_SUCCESS != cudppPlan(theCudpp, &scanPlan, config, _params.width, _params.height, d_satPitchInElements))
    fprintf(stderr, "Error creating CUDPPPlan.\n");

  // scan rows
  cudppMultiScan(scanPlan, _data.SATs[0], _data.SATs[0], _params.width, _params.height);

  // transpose so columns become rows
  d_transpose<T, 16> <<<blocks_2, threads_2, 0>>>(_data.SATs[1], _data.SATs[0],
                                                      d_satPitchInElements_T,
                                                      d_satPitchInElements,
                                                      _params.width,
                                                      _params.height);

  if (CUDPP_SUCCESS != cudppDestroyPlan(scanPlan))
    fprintf(stderr, "Error destroying CUDPPPlan.\n");
  if (CUDPP_SUCCESS != cudppPlan(theCudpp, &scanPlan, config, _params.height, _params.width, d_satPitchInElements_T))
    fprintf(stderr, "Error creating CUDPPPlan.\n");
  // scan columns
  cudppMultiScan(scanPlan, _data.SATs[1], _data.SATs[1], _params.height, _params.width);

  // transpose back
  d_transpose<T, 16> <<<blocks_2_T, threads_2, 0>>>(_data.SATs[0], _data.SATs[1],
                                                        d_satPitchInElements,
                                                        d_satPitchInElements_T,
                                                        _params.height,
                                                        _params.width);

  if (CUDPP_SUCCESS != cudppDestroyPlan(scanPlan))
    fprintf(stderr, "Error destroying CUDPPPlan.\n");
  d_blur_step2<<<blocks_2, threads_2>>>(_target, _backBuffer, _data.SATs[0],
                                        d_satPitchInElements, _source, _params);
}




template
void init_buffer<float>(DataMc<float>&, const Parameters<float>&, bool, int);
template float launch_kernel(hipGraphicsResource* dst,
                             DataMc<float>& ddata,
                             const Parameters<float>& params,
                             bool advance,
                             int direction_mode);
template
void cleanup_cuda(DataMc<float>& ddata);